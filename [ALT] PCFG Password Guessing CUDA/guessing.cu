#include "hip/hip_runtime.h"
#include "PCFG.h"
using namespace std;

void PriorityQueue::CalProb(PT &pt) {
    // 计算PriorityQueue里面一个PT的流程如下：
    // 1. 首先需要计算一个PT本身的概率。例如，L6S1的概率为0.15
    // 2. 需要注意的是，Queue里面的PT不是“纯粹的”PT，而是除了最后一个segment以外，全部被value实例化的PT
    // 3. 所以，对于L6S1而言，其在Queue里面的实际PT可能是123456S1，其中“123456”为L6的一个具体value。
    // 4. 这个时候就需要计算123456在L6中出现的概率了。假设123456在所有L6 segment中的概率为0.1，那么123456S1的概率就是0.1*0.15

    // 计算一个PT本身的概率。后续所有具体segment value的概率，直接累乘在这个初始概率值上
    pt.prob = pt.preterm_prob;

    // index: 标注当前segment在PT中的位置
    int index = 0;


    for (int idx : pt.curr_indices)
    {
        // pt.content[index].PrintSeg();
        if (pt.content[index].type == 1)
        {
            // 下面这行代码的意义：
            // pt.content[index]：目前需要计算概率的segment
            // m.FindLetter(seg): 找到一个letter segment在模型中的对应下标
            // m.letters[m.FindLetter(seg)]：一个letter segment在模型中对应的所有统计数据
            // m.letters[m.FindLetter(seg)].ordered_values：一个letter segment在模型中，所有value的总数目
            pt.prob *= m.letters[m.FindLetter(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.letters[m.FindLetter(pt.content[index])].total_freq;
            // cout << m.letters[m.FindLetter(pt.content[index])].ordered_freqs[idx] << endl;
            // cout << m.letters[m.FindLetter(pt.content[index])].total_freq << endl;
        }
        if (pt.content[index].type == 2)
        {
            pt.prob *= m.digits[m.FindDigit(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.digits[m.FindDigit(pt.content[index])].total_freq;
            // cout << m.digits[m.FindDigit(pt.content[index])].ordered_freqs[idx] << endl;
            // cout << m.digits[m.FindDigit(pt.content[index])].total_freq << endl;
        }
        if (pt.content[index].type == 3)
        {
            pt.prob *= m.symbols[m.FindSymbol(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.symbols[m.FindSymbol(pt.content[index])].total_freq;
            // cout << m.symbols[m.FindSymbol(pt.content[index])].ordered_freqs[idx] << endl;
            // cout << m.symbols[m.FindSymbol(pt.content[index])].total_freq << endl;
        }
        index += 1;
    }
    // cout << pt.prob << endl;
}

void PriorityQueue::init() {
    // cout << m.ordered_pts.size() << endl;
    // 用所有可能的PT，按概率降序填满整个优先队列
    for (PT pt : m.ordered_pts)
    {
        for (segment seg : pt.content)
        {
            if (seg.type == 1)
            {
                // 下面这行代码的意义：
                // max_indices用来表示PT中各个segment的可能数目。例如，L6S1中，假设模型统计到了100个L6，那么L6对应的最大下标就是99
                // （但由于后面采用了"<"的比较关系，所以其实max_indices[0]=100）
                // m.FindLetter(seg): 找到一个letter segment在模型中的对应下标
                // m.letters[m.FindLetter(seg)]：一个letter segment在模型中对应的所有统计数据
                // m.letters[m.FindLetter(seg)].ordered_values：一个letter segment在模型中，所有value的总数目
                pt.max_indices.emplace_back(m.letters[m.FindLetter(seg)].ordered_values.size());
            }
            if (seg.type == 2)
            {
                pt.max_indices.emplace_back(m.digits[m.FindDigit(seg)].ordered_values.size());
            }
            if (seg.type == 3)
            {
                pt.max_indices.emplace_back(m.symbols[m.FindSymbol(seg)].ordered_values.size());
            }
        }
        pt.preterm_prob = float(m.preterm_freq[m.FindPT(pt)]) / m.total_preterm;
        // pt.PrintPT();
        // cout << " " << m.preterm_freq[m.FindPT(pt)] << " " << m.total_preterm << " " << pt.preterm_prob << endl;

        // 计算当前pt的概率
        CalProb(pt);
        // 将PT放入优先队列
        priority.emplace_back(pt);
    }
    // cout << "priority size:" << priority.size() << endl;
}

void PriorityQueue::PopNext() {
    // 对优先队列最前面的PT，首先利用这个PT生成一系列猜测
    // <=== 串行方法 ===>
    // Generate(priority.front());
    // <=== CUDA方法 ===>
    CUDAGenerate(priority.front());

    // 然后需要根据即将出队的PT，生成一系列新的PT
    vector<PT> new_pts = priority.front().NewPTs();
    for (PT pt : new_pts)
    {
        // 计算概率
        CalProb(pt);
        // 接下来的这个循环，作用是根据概率，将新的PT插入到优先队列中
        for (auto iter = priority.begin(); iter != priority.end(); iter++)
        {
            // 对于非队首和队尾的特殊情况
            if (iter != priority.end() - 1 && iter != priority.begin())
            {
                // 判定概率
                if (pt.prob <= iter->prob && pt.prob > (iter + 1)->prob)
                {
                    priority.emplace(iter + 1, pt);
                    break;
                }
            }
            if (iter == priority.end() - 1)
            {
                priority.emplace_back(pt);
                break;
            }
            if (iter == priority.begin() && iter->prob < pt.prob)
            {
                priority.emplace(iter, pt);
                break;
            }
        }
    }

    // 现在队首的PT善后工作已经结束，将其出队（删除）
    priority.erase(priority.begin());
}

// 这个函数你就算看不懂，对并行算法的实现影响也不大
// 当然如果你想做一个基于多优先队列的并行算法，可能得稍微看一看了
vector<PT> PT::NewPTs() {
    // 存储生成的新PT
    vector<PT> res;

    // 假如这个PT只有一个segment
    // 那么这个segment的所有value在出队前就已经被遍历完毕，并作为猜测输出
    // 因此，所有这个PT可能对应的口令猜测已经遍历完成，无需生成新的PT
    if (content.size() == 1)
    {
        return res;
    }
    else
    {
        // 最初的pivot值。我们将更改位置下标大于等于这个pivot值的segment的值（最后一个segment除外），并且一次只更改一个segment
        // 上面这句话里是不是有没看懂的地方？接着往下看你应该会更明白
        int init_pivot = pivot;

        // 开始遍历所有位置值大于等于init_pivot值的segment
        // 注意i < curr_indices.size() - 1，也就是除去了最后一个segment（这个segment的赋值预留给并行环节）
        for (int i = pivot; i < curr_indices.size() - 1; i += 1)
        {
            // curr_indices: 标记各segment目前的value在模型里对应的下标
            curr_indices[i] += 1;

            // max_indices：标记各segment在模型中一共有多少个value
            if (curr_indices[i] < max_indices[i])
            {
                // 更新pivot值
                pivot = i;
                res.emplace_back(*this);
            }

            // 这个步骤对于你理解pivot的作用、新PT生成的过程而言，至关重要
            curr_indices[i] -= 1;
        }
        pivot = init_pivot;
        return res;
    }

    return res;
}

// 这个函数是PCFG并行化算法的主要载体
// 尽量看懂，然后进行并行实现
void PriorityQueue::Generate(PT pt) {
    // 计算PT的概率，这里主要是给PT的概率进行初始化
    CalProb(pt);

    // 对于只有一个segment的PT，直接遍历生成其中的所有value即可
    if (pt.content.size() == 1)
    {
        // 指向最后一个segment的指针，这个指针实际指向模型中的统计数据
        segment *a;
        // 在模型中定位到这个segment
        if (pt.content[0].type == 1)
        {
            a = &m.letters[m.FindLetter(pt.content[0])];
        }
        if (pt.content[0].type == 2)
        {
            a = &m.digits[m.FindDigit(pt.content[0])];
        }
        if (pt.content[0].type == 3)
        {
            a = &m.symbols[m.FindSymbol(pt.content[0])];
        }

        // Multi-thread TODO：
        // 这个for循环就是你需要进行并行化的主要部分了，特别是在多线程&GPU编程任务中
        // 可以看到，这个循环本质上就是把模型中一个segment的所有value，赋值到PT中，形成一系列新的猜测
        // 这个过程是可以高度并行化的
        for (int i = 0; i < pt.max_indices[0]; i += 1)
        {
            string guess = a->ordered_values[i];
            // cout << guess << endl;
            guesses.emplace_back(guess);
            total_guesses += 1;
        }
    }
    else
    {
        string guess;
        int seg_idx = 0;
        // 这个for循环的作用：给当前PT的所有segment赋予实际的值（最后一个segment除外）
        // segment值根据curr_indices中对应的值加以确定
        // 这个for循环你看不懂也没太大问题，并行算法不涉及这里的加速
        for (int idx : pt.curr_indices)
        {
            if (pt.content[seg_idx].type == 1)
            {
                guess += m.letters[m.FindLetter(pt.content[seg_idx])].ordered_values[idx];
            }
            if (pt.content[seg_idx].type == 2)
            {
                guess += m.digits[m.FindDigit(pt.content[seg_idx])].ordered_values[idx];
            }
            if (pt.content[seg_idx].type == 3)
            {
                guess += m.symbols[m.FindSymbol(pt.content[seg_idx])].ordered_values[idx];
            }
            seg_idx += 1;
            if (seg_idx == pt.content.size() - 1)
            {
                break;
            }
        }

        // 指向最后一个segment的指针，这个指针实际指向模型中的统计数据
        segment *a;
        if (pt.content[pt.content.size() - 1].type == 1)
        {
            a = &m.letters[m.FindLetter(pt.content[pt.content.size() - 1])];
        }
        if (pt.content[pt.content.size() - 1].type == 2)
        {
            a = &m.digits[m.FindDigit(pt.content[pt.content.size() - 1])];
        }
        if (pt.content[pt.content.size() - 1].type == 3)
        {
            a = &m.symbols[m.FindSymbol(pt.content[pt.content.size() - 1])];
        }

        // Multi-thread TODO：
        // 这个for循环就是你需要进行并行化的主要部分了，特别是在多线程&GPU编程任务中
        // 可以看到，这个循环本质上就是把模型中一个segment的所有value，赋值到PT中，形成一系列新的猜测
        // 这个过程是可以高度并行化的
        for (int i = 0; i < pt.max_indices[pt.content.size() - 1]; i += 1)
        {
            string temp = guess + a->ordered_values[i];
            // cout << temp << endl;
            guesses.emplace_back(temp);
            total_guesses += 1;
        }
    }
}

// <================ CUDA 实现 ================>

// 初始化
void PriorityQueue::InitCudaBuffers() {
    d_capacity = MAX_N;
    hipMalloc(&d_values, d_capacity * MAX_STR_LEN);
    hipMalloc(&d_output, d_capacity * MAX_GUESS_LEN);
    hipMalloc(&d_prefix, MAX_STR_LEN);
}

// 释放
void PriorityQueue::FreeCudaBuffers() {
    if (d_values) hipFree(d_values);
    if (d_output) hipFree(d_output);
    if (d_prefix) hipFree(d_prefix);
    d_values = d_output = d_prefix = nullptr;
    d_capacity = 0;
}

// CPU -> GPU
void PriorityQueue::LoadAllOrderedValuesToGPU() {
    for (auto &seg : m.letters) total_letter_count += seg.ordered_values.size();
    for (auto &seg : m.digits)  total_digit_count += seg.ordered_values.size();
    for (auto &seg : m.symbols) total_symbol_count += seg.ordered_values.size();

    char *h_letters_all = new char[total_letter_count * MAX_STR_LEN]();
    char *h_digits_all  = new char[total_digit_count * MAX_STR_LEN]();
    char *h_symbols_all = new char[total_symbol_count * MAX_STR_LEN]();

    vector<int> h_letter_offsets, h_digit_offsets, h_symbol_offsets;
    int offset = 0;
    for (auto &seg : m.letters) {
        h_letter_offsets.push_back(offset);
        for (auto &v : seg.ordered_values) {
            strncpy(h_letters_all + offset * MAX_STR_LEN, v.c_str(), MAX_STR_LEN - 1);
            offset++;
        }
    }
    offset = 0;
    for (auto &seg : m.digits) {
        h_digit_offsets.push_back(offset);
        for (auto &v : seg.ordered_values) {
            strncpy(h_digits_all + offset * MAX_STR_LEN, v.c_str(), MAX_STR_LEN - 1);
            offset++;
        }
    }
    offset = 0;
    for (auto &seg : m.symbols) {
        h_symbol_offsets.push_back(offset);
        for (auto &v : seg.ordered_values) {
            strncpy(h_symbols_all + offset * MAX_STR_LEN, v.c_str(), MAX_STR_LEN - 1);
            offset++;
        }
    }

    hipMalloc(&d_letters_all, total_letter_count * MAX_STR_LEN);
    hipMalloc(&d_digits_all,  total_digit_count  * MAX_STR_LEN);
    hipMalloc(&d_symbols_all, total_symbol_count * MAX_STR_LEN);
    hipMemcpy(d_letters_all, h_letters_all, total_letter_count * MAX_STR_LEN, hipMemcpyHostToDevice);
    hipMemcpy(d_digits_all,  h_digits_all,  total_digit_count  * MAX_STR_LEN, hipMemcpyHostToDevice);
    hipMemcpy(d_symbols_all, h_symbols_all, total_symbol_count * MAX_STR_LEN, hipMemcpyHostToDevice);

    hipMalloc(&d_letter_offsets, h_letter_offsets.size() * sizeof(int));
    hipMalloc(&d_digit_offsets,  h_digit_offsets.size()  * sizeof(int));
    hipMalloc(&d_symbol_offsets, h_symbol_offsets.size() * sizeof(int));
    hipMemcpy(d_letter_offsets, h_letter_offsets.data(), h_letter_offsets.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_digit_offsets,  h_digit_offsets.data(),  h_digit_offsets.size()  * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_symbol_offsets, h_symbol_offsets.data(), h_symbol_offsets.size() * sizeof(int), hipMemcpyHostToDevice);

    // 缓存在 host 端
    h_letter_offsets_gpu = h_letter_offsets;
    h_digit_offsets_gpu  = h_digit_offsets;
    h_symbol_offsets_gpu = h_symbol_offsets;

    delete[] h_letters_all;
    delete[] h_digits_all;
    delete[] h_symbols_all;
}

// 释放
void PriorityQueue::FreeGlobalBuffers() {
    hipFree(d_letters_all); hipFree(d_digits_all); hipFree(d_symbols_all);
    hipFree(d_letter_offsets); hipFree(d_digit_offsets); hipFree(d_symbol_offsets);
}

__global__ void generate_kernel_indexed(char *d_values_all, int base_offset, int value_count, char *d_output, int max_len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= value_count) return;
    char *src = d_values_all + (base_offset + idx) * max_len;
    char *dst = d_output + idx * max_len;
    for (int i = 0; i < max_len; i += 16) {
        int4 data = *((int4 *)(src + i));
        *((int4 *)(dst + i)) = data;
    }
}

__global__ void generate_kernel_indexed_concat(
    char *prefix, int prefix_len,
    char *d_values_all, int base_offset,
    int value_count, char *d_output, int max_suffix_len
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= value_count) return;
    char *src = d_values_all + (base_offset + idx) * max_suffix_len;
    char *dst = d_output + idx * MAX_GUESS_LEN;
    for (int i = 0; i < prefix_len; ++i)
        dst[i] = prefix[i];
    for (int i = 0; i < max_suffix_len; i += 16) {
        int4 data = *((int4 *)(src + i));
        *((int4 *)(dst + prefix_len + i)) = data;
    }
}

void PriorityQueue::CUDAGenerate(PT pt) {
    CalProb(pt);

    if (pt.content.size() == 1) {
        segment *a;
        char *d_pool = nullptr;
        int seg_idx = -1;

        if (pt.content[0].type == 1) {
            seg_idx = m.FindLetter(pt.content[0]);
            d_pool = d_letters_all;
        } else if (pt.content[0].type == 2) {
            seg_idx = m.FindDigit(pt.content[0]);
            d_pool = d_digits_all;
        } else {
            seg_idx = m.FindSymbol(pt.content[0]);
            d_pool = d_symbols_all;
        }

        int base_offset = (pt.content[0].type == 1) ? h_letter_offsets_gpu[seg_idx] :
                          (pt.content[0].type == 2) ? h_digit_offsets_gpu[seg_idx] :
                                                      h_symbol_offsets_gpu[seg_idx];

        a = (pt.content[0].type == 1) ? &m.letters[seg_idx] :
            (pt.content[0].type == 2) ? &m.digits[seg_idx] :
                                       &m.symbols[seg_idx];

        int N = pt.max_indices[0];
        if (N < 100000) {
            for (int i = 0; i < N; ++i)
                guesses.push_back(a->ordered_values[i]);
            total_guesses += N;
            return;
        }
        if (N > d_capacity) {
            std::cerr << "CUDAGenerate error: batch size " << N << " exceeds GPU buffer capacity " << d_capacity << "\n";
            return;
        }

        int blockSize = 256;
        int numBlocks = (N + blockSize - 1) / blockSize;
        generate_kernel_indexed<<<numBlocks, blockSize>>>(d_pool, base_offset, N, d_output, MAX_STR_LEN);
        hipDeviceSynchronize();

        char *h_output = new char[N * MAX_STR_LEN]();
        hipMemcpy(h_output, d_output, N * MAX_STR_LEN, hipMemcpyDeviceToHost);
        for (int i = 0; i < N; ++i)
            guesses.emplace_back(h_output + i * MAX_STR_LEN);
        total_guesses += N;
        delete[] h_output;
    } else {
        // 多段
        string prefix;
        for (int seg_idx = 0; seg_idx < pt.content.size() - 1; ++seg_idx) {
            int idx = pt.curr_indices[seg_idx];
            if (pt.content[seg_idx].type == 1)
                prefix += m.letters[m.FindLetter(pt.content[seg_idx])].ordered_values[idx];
            else if (pt.content[seg_idx].type == 2)
                prefix += m.digits[m.FindDigit(pt.content[seg_idx])].ordered_values[idx];
            else
                prefix += m.symbols[m.FindSymbol(pt.content[seg_idx])].ordered_values[idx];
        }

        int seg_idx = -1;
        char *d_pool = nullptr;

        if (pt.content.back().type == 1) {
            seg_idx = m.FindLetter(pt.content.back());
            d_pool = d_letters_all;
        } else if (pt.content.back().type == 2) {
            seg_idx = m.FindDigit(pt.content.back());
            d_pool = d_digits_all;
        } else {
            seg_idx = m.FindSymbol(pt.content.back());
            d_pool = d_symbols_all;
        }

        int base_offset = (pt.content.back().type == 1) ? h_letter_offsets_gpu[seg_idx] :
                          (pt.content.back().type == 2) ? h_digit_offsets_gpu[seg_idx] :
                                                          h_symbol_offsets_gpu[seg_idx];

        segment *a = (pt.content.back().type == 1) ? &m.letters[seg_idx] :
                       (pt.content.back().type == 2) ? &m.digits[seg_idx] :
                                                       &m.symbols[seg_idx];

        int N = pt.max_indices.back();
        if (N < 100000) {
            for (int i = 0; i < N; ++i)
                guesses.push_back(prefix + a->ordered_values[i]);
            total_guesses += N;
            return;
        }
        if (N > d_capacity) {
            std::cerr << "CUDAGenerate error: batch size " << N << " exceeds GPU buffer capacity " << d_capacity << "\n";
            return;
        }

        char h_prefix[MAX_STR_LEN] = {};
        strncpy(h_prefix, prefix.c_str(), MAX_STR_LEN - 1);
        hipMemcpy(d_prefix, h_prefix, MAX_STR_LEN, hipMemcpyHostToDevice);

        int blockSize = 256;
        int numBlocks = (N + blockSize - 1) / blockSize;
        generate_kernel_indexed_concat<<<numBlocks, blockSize>>>(
            d_prefix, prefix.length(), d_pool, base_offset, N, d_output, MAX_STR_LEN);
        hipDeviceSynchronize();

        char *h_output = new char[N * MAX_GUESS_LEN]();
        hipMemcpy(h_output, d_output, N * MAX_GUESS_LEN, hipMemcpyDeviceToHost);
        for (int i = 0; i < N; ++i)
            guesses.emplace_back(h_output + i * MAX_GUESS_LEN);
        total_guesses += N;
        delete[] h_output;
    }
}

// 核函数：每个线程负责一个PT的一个猜测
__global__ void generate_kernel_multi_pt(
    char *d_prefixes,           // batch_size * MAX_STR_LEN， 每个PT的prefix字符串
    int *d_last_segment_types,  // batch_size，最后segment类型
    int *d_last_segment_offsets,// batch_size，最后segment字符串的base offset
    int *d_last_segment_max_indices, // batch_size，最后segmentmax_indices
    int *d_pt_indices,          // total_guess_count，映射每个猜测属于哪个PT
    int *d_value_indices,       // total_guess_count，映射每个猜测在最后segment的哪个值
    char *d_letters_all,
    char *d_digits_all,
    char *d_symbols_all,
    char *d_output,
    int total_guess_count
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_guess_count) return;

    int pt_idx = d_pt_indices[idx];
    int value_idx = d_value_indices[idx];

    // 计算prefix长度 - 修复：使用更安全的长度计算
    int prefix_len = 0;
    for (int i = 0; i < MAX_STR_LEN - 1; ++i) {  // 保留空间给'\0'
        if (d_prefixes[pt_idx * MAX_STR_LEN + i] == '\0') {
            prefix_len = i;
            break;
        }
        prefix_len = i + 1;  // 如果没有找到'\0'，则使用实际长度
    }

    // 输出指针
    char *dst = d_output + idx * MAX_GUESS_LEN;
    
    // 初始化输出缓冲区 - 修复：确保清零
    for (int i = 0; i < MAX_GUESS_LEN; ++i) {
        dst[i] = '\0';
    }

    // 拷贝 prefix
    for (int i = 0; i < prefix_len && i < MAX_GUESS_LEN - 1; ++i) {
        dst[i] = d_prefixes[pt_idx * MAX_STR_LEN + i];
    }

    // 根据最后segment类型，选对应的字符串池
    int seg_type = d_last_segment_types[pt_idx];
    int base_offset = d_last_segment_offsets[pt_idx];
    char *pool = nullptr;

    if (seg_type == 1) {
        pool = d_letters_all;
    } else if (seg_type == 2) {
        pool = d_digits_all;
    } else {
        pool = d_symbols_all;
    }

    // 修复：检查越界和安全拷贝
    if (pool != nullptr && value_idx >= 0) {
        char *src = pool + (base_offset + value_idx) * MAX_STR_LEN;  // 使用MAX_STR_LEN而不是max_suffix_len
        
        // 安全拷贝最后segment字符串
        int remaining_space = MAX_GUESS_LEN - prefix_len - 1;  // 保留空间给'\0'
        for (int i = 0; i < MAX_STR_LEN && i < remaining_space; ++i) {
            if (src[i] == '\0') break;  // 遇到字符串结束符就停止
            dst[prefix_len + i] = src[i];
        }
    }
}

// 批量处理函数
void PriorityQueue::CUDAPopNext() {
    int batch_size = BATCH_SIZE;
    if (batch_size > (int)priority.size()) batch_size = priority.size();
    if (batch_size == 0) return;

    // 复制批量PT
    vector<PT> batch_pts(priority.begin(), priority.begin() + batch_size);

    // 计算每个PT最后segment max_indices总和（总猜测数）
    int total_guess_count = 0;
    vector<int> last_seg_max_indices(batch_size);
    for (int i = 0; i < batch_size; ++i) {
        if (batch_pts[i].max_indices.empty()) {
            std::cerr << "Error: PT " << i << " has empty max_indices" << std::endl;
            return;
        }
        last_seg_max_indices[i] = batch_pts[i].max_indices.back();
        total_guess_count += last_seg_max_indices[i];
    }
    
    if (total_guess_count == 0) {
        std::cerr << "Warning: No guesses to generate" << std::endl;
        return;
    }
    
    if (total_guess_count > d_capacity) {
        std::cerr << "Batch total guesses " << total_guess_count << " exceed GPU capacity " << d_capacity << std::endl;
        return;
    }

    // Host端构造 prefix 字符串数组 - 修复：确保正确的字符串构造
    char *h_prefixes = new char[batch_size * MAX_STR_LEN]();
    for (int i = 0; i < batch_size; ++i) {
        string prefix;
        
        // 只处理前面的segment，最后一个segment在GPU上处理
        int num_prefix_segments = (int)batch_pts[i].content.size() - 1;
        for (int seg_idx = 0; seg_idx < num_prefix_segments; ++seg_idx) {
            if (seg_idx >= (int)batch_pts[i].curr_indices.size()) {
                std::cerr << "Error: curr_indices size mismatch for PT " << i << std::endl;
                break;
            }
            
            int idx = batch_pts[i].curr_indices[seg_idx];
            int seg_type = batch_pts[i].content[seg_idx].type;
            
            // 修复：添加边界检查
            if (seg_type == 1) {
                int letter_idx = m.FindLetter(batch_pts[i].content[seg_idx]);
                if (letter_idx >= 0 && letter_idx < (int)m.letters.size() && 
                    idx >= 0 && idx < (int)m.letters[letter_idx].ordered_values.size()) {
                    prefix += m.letters[letter_idx].ordered_values[idx];
                }
            } else if (seg_type == 2) {
                int digit_idx = m.FindDigit(batch_pts[i].content[seg_idx]);
                if (digit_idx >= 0 && digit_idx < (int)m.digits.size() && 
                    idx >= 0 && idx < (int)m.digits[digit_idx].ordered_values.size()) {
                    prefix += m.digits[digit_idx].ordered_values[idx];
                }
            } else {
                int symbol_idx = m.FindSymbol(batch_pts[i].content[seg_idx]);
                if (symbol_idx >= 0 && symbol_idx < (int)m.symbols.size() && 
                    idx >= 0 && idx < (int)m.symbols[symbol_idx].ordered_values.size()) {
                    prefix += m.symbols[symbol_idx].ordered_values[idx];
                }
            }
        }
        
        // 安全拷贝prefix
        size_t copy_len = min(prefix.length(), (size_t)(MAX_STR_LEN - 1));
        strncpy(h_prefixes + i * MAX_STR_LEN, prefix.c_str(), copy_len);
        h_prefixes[i * MAX_STR_LEN + copy_len] = '\0';  // 确保以'\0'结尾
    }

    // Host端准备最后segment类型数组和offset数组 - 修复：添加错误检查
    int *h_last_segment_types = new int[batch_size];
    int *h_last_segment_offsets = new int[batch_size];
    int *h_last_segment_max_indices = new int[batch_size];

    for (int i = 0; i < batch_size; ++i) {        
        int seg_type = batch_pts[i].content.back().type;
        h_last_segment_types[i] = seg_type;
        h_last_segment_max_indices[i] = batch_pts[i].max_indices.back();

        int seg_idx = -1;
        if (seg_type == 1) {
            seg_idx = m.FindLetter(batch_pts[i].content.back());
        } else if (seg_type == 2) {
            seg_idx = m.FindDigit(batch_pts[i].content.back());
        } else {
            seg_idx = m.FindSymbol(batch_pts[i].content.back());
        }

        // 计算偏移 - 修复：添加边界检查
        int offset = 0;
        if (seg_type == 1 && seg_idx < (int)h_letter_offsets_gpu.size()) {
            offset = h_letter_offsets_gpu[seg_idx];
        } else if (seg_type == 2 && seg_idx < (int)h_digit_offsets_gpu.size()) {
            offset = h_digit_offsets_gpu[seg_idx];
        } else if (seg_type == 3 && seg_idx < (int)h_symbol_offsets_gpu.size()) {
            offset = h_symbol_offsets_gpu[seg_idx];
        }

        h_last_segment_offsets[i] = offset;
    }

    // 为每个猜测准备PT索引和最后segment的value索引数组
    int *h_pt_indices = new int[total_guess_count];
    int *h_value_indices = new int[total_guess_count];

    int pos = 0;
    for (int i = 0; i < batch_size; ++i) {
        for (int v_idx = 0; v_idx < last_seg_max_indices[i]; ++v_idx) {
            h_pt_indices[pos] = i;
            h_value_indices[pos] = v_idx;
            pos++;
        }
    }

    // GPU内存分配和复制 - 修复：避免goto跳过变量初始化
    char *d_prefixes = nullptr;
    int *d_last_segment_types = nullptr;
    int *d_last_segment_offsets = nullptr;
    int *d_last_segment_max_indices = nullptr;
    int *d_pt_indices = nullptr;
    int *d_value_indices = nullptr;
    
    // 预先声明所有变量避免goto跳过初始化
    int blockSize = 256;
    int numBlocks = (total_guess_count + blockSize - 1) / blockSize;
    char *h_output = nullptr;
    int valid_guesses = 0;

    // GPU内存分配
    do {
        hipMalloc(&d_prefixes, batch_size * MAX_STR_LEN);
        hipMalloc(&d_last_segment_types, batch_size * sizeof(int));
        hipMalloc(&d_last_segment_offsets, batch_size * sizeof(int));
        hipMalloc(&d_last_segment_max_indices, batch_size * sizeof(int));
        hipMalloc(&d_pt_indices, total_guess_count * sizeof(int));
        hipMalloc(&d_value_indices, total_guess_count * sizeof(int));

        // 拷贝数据到GPU
        hipMemcpy(d_prefixes, h_prefixes, batch_size * MAX_STR_LEN, hipMemcpyHostToDevice);
        hipMemcpy(d_last_segment_types, h_last_segment_types, batch_size * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_last_segment_offsets, h_last_segment_offsets, batch_size * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_last_segment_max_indices, h_last_segment_max_indices, batch_size * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_pt_indices, h_pt_indices, total_guess_count * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_value_indices, h_value_indices, total_guess_count * sizeof(int), hipMemcpyHostToDevice);

        // 分配GPU输出缓冲区
        if (d_capacity < total_guess_count) {
            if (d_output) hipFree(d_output);
            hipMalloc(&d_output, total_guess_count * MAX_GUESS_LEN);
            d_capacity = total_guess_count;
        }

        // 调用核函数
        generate_kernel_multi_pt<<<numBlocks, blockSize>>>(
            d_prefixes,
            d_last_segment_types,
            d_last_segment_offsets,
            d_last_segment_max_indices,
            d_pt_indices,
            d_value_indices,
            d_letters_all,
            d_digits_all,
            d_symbols_all,
            d_output,
            total_guess_count
        );

        // 拷贝结果回Host并保存
        h_output = new char[total_guess_count * MAX_GUESS_LEN]();
        hipMemcpy(h_output, d_output, total_guess_count * MAX_GUESS_LEN, hipMemcpyDeviceToHost);
        
        // 检查生成的猜测是否有效
        for (int i = 0; i < total_guess_count; ++i) {
            char* guess_ptr = h_output + i * MAX_GUESS_LEN;
            if (strlen(guess_ptr) > 0) {  // 只添加非空猜测
                guesses.emplace_back(guess_ptr);
                valid_guesses++;
            }
        }
        total_guesses += valid_guesses;
                
    } while (false);  // 只执行一次的do-while循环，用于替代goto

    // 清理Host内存
    delete[] h_prefixes;
    delete[] h_last_segment_types;
    delete[] h_last_segment_offsets;
    delete[] h_last_segment_max_indices;
    delete[] h_pt_indices;
    delete[] h_value_indices;
    
    if (h_output) {
        delete[] h_output;
    }

    // 释放GPU内存
    if (d_prefixes) hipFree(d_prefixes);
    if (d_last_segment_types) hipFree(d_last_segment_types);
    if (d_last_segment_offsets) hipFree(d_last_segment_offsets);
    if (d_last_segment_max_indices) hipFree(d_last_segment_max_indices);
    if (d_pt_indices) hipFree(d_pt_indices);
    if (d_value_indices) hipFree(d_value_indices);

    // 删除优先队列中处理的PT
    priority.erase(priority.begin(), priority.begin() + batch_size);
}

// <==========================================>